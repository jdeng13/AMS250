#include "hip/hip_runtime.h"
// Matrix multiplication kernel
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
// Block row and column
int blockRow = blockIdx.y;
int blockCol = blockIdx.x;
// Each thread block computes one sub-matrix of C
Matrix Csub = GetSubMatrix(C, blockRow, blockCol);
// Each thread computes one elements of Csub
float Cvalue = 0;
// Thread row and column with Csub
int row = threadIdx.y;
int col = threadIdx.y;
